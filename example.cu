#include <hip/hip_runtime.h>
#include <thrust/version.h>

#include <cusp/version.h>
#include <cusp/hyb_matrix.h>
#include <cusp/io/matrix_market.h>
#include <cusp/krylov/cg.h>

#include <iostream>

int main(void)
{
    int cuda_major =  CUDA_VERSION / 1000;
    int cuda_minor = (CUDA_VERSION % 1000) / 10;
    int thrust_major = THRUST_MAJOR_VERSION;
    int thrust_minor = THRUST_MINOR_VERSION;
    int cusp_major = CUSP_MAJOR_VERSION;
    int cusp_minor = CUSP_MINOR_VERSION;
    std::cout << "CUDA   v" << cuda_major   << "." << cuda_minor   << std::endl;
    std::cout << "Thrust v" << thrust_major << "." << thrust_minor << std::endl;
    std::cout << "Cusp   v" << cusp_major   << "." << cusp_minor   << std::endl;

    // create an empty sparse matrix structure (HYB format)
    cusp::hyb_matrix<int, float, cusp::device_memory> A;

    // load a matrix stored in Matrix-Market format
    cusp::io::read_matrix_market_file(A, "./testing/data/laplacian/5pt_10x10.mtx");

    // allocate storage for solution (x) and right hand side (b)
    cusp::array1d<float, cusp::device_memory> x(A.num_rows, 0);
    cusp::array1d<float, cusp::device_memory> b(A.num_rows, 1);

    // solve the linear system A * x = b with the conjugate gradient method
    cusp::krylov::cg(A, x, b);

    return 0;
}
